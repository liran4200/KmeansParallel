#include "hip/hip_runtime.h"
#include "header.h"

__global__ void  findNeasrstClusterCenter(int blockNums, int k, Point *points, Cluster *clusters, boolean *isChange)
{
	
	int j;
	int indexPoints = threadIdx.x + blockIdx.x * blockNums;
	double currentDistance = 0;
	int minIndex = 0; // save min index in the array
	double minDistance = cudaDistance(points[indexPoints], clusters[0]);
	for (j = 1; j < k; j++)
	{
		currentDistance = cudaDistance(points[indexPoints], clusters[j]);
		if (currentDistance < minDistance)
		{
			minDistance = currentDistance;
			minIndex = j;
		}
	}

	//enough one point that moved to chagne it to TRUE
	if (points[indexPoints].clusterId != clusters[minIndex].id)
		*isChange = TRUE;

	points[indexPoints].clusterId = clusters[minIndex].id;
	
}

boolean classifyEachPointToClustereCentersWithCuda(int n, int k, Point *arrPoints, Cluster *arrClusters)
{
	boolean *dev_isChanged;
	boolean isChange = FALSE;
	hipError_t cudaStatus;
	hipDeviceProp_t props;
	int blockNums, threadNum;
	hipGetDeviceProperties(&props, 0);

	// calculate the number of blocks required in this computer
	blockNums = 1 + n /props.maxThreadsPerBlock;

	// Allocate GPU arrays - arrPoints, arrClusters.
	cudaStatus = hipMalloc((void**)&dev_isChanged, sizeof(int));
	checkCudaStatus(cudaStatus, "hipMalloc failed!");

	cudaStatus = hipMemcpy(dev_isChanged, &isChange, sizeof(int), hipMemcpyHostToDevice);
	checkCudaStatus(cudaStatus, "hipMemcpy failed!");

	threadNum = n / blockNums;

	// Launch a kernel on the GPU with one thread for each element.
	findNeasrstClusterCenter << < blockNums, threadNum >> >(blockNums, k, arrPoints, arrClusters, dev_isChanged);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	//cudaStatus = hipDeviceSynchronize();
	//checkCudaStatus(cudaStatus, "hipDeviceSynchronize returned error code");

	cudaStatus = hipMemcpy(&isChange, dev_isChanged, sizeof(int), hipMemcpyDeviceToHost);
	checkCudaStatus(cudaStatus, "hipMalloc failed!");

	hipFree(dev_isChanged);

	return isChange;
}

void checkCudaStatus(hipError_t e, const char *message)
{
	// check if the cuda status was ok
	if (e != hipSuccess)
	{
		printf(message);
		fflush(stdout);
		exit(1);
	}
}

__device__ double cudaDistance(Point p1, Cluster c1)
{
	double deltaX = pow(p1.x - c1.x, 2);
	double deltaY = pow(p1.y - c1.y, 2);
	return sqrt(deltaX + deltaY);
}

